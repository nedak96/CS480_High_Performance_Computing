
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
using namespace std;

//runs the neural network
__global__
void forward(float *inp, unsigned char* labels, float *weights, float *bias, float *correct){
	int ans;
	*correct = 0;
	float out[10];
	int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int n = index; n < 10000; n += stride){
		ans = 0;
		for(int i=0; i<10; i++){
		 out[i] = 0;
		 for(int j=0; j<784; j++){
			 out[i] += weights[i*784+j]*inp[n*784+j];
		 }
		 out[i] += bias[i];
		 if(out[i] > out[ans]) ans = i;
		}
		if(ans == (int)labels[n]) atomicAdd(correct,1);
	}
}

//trains the neural network
__global__
void train(float *inp, unsigned char *labels, int *shuffled, float *wd, float *bd, float *weights, float *bias){
	int index = threadIdx.x;
  int stride = blockDim.x;
	for(int z=index; z<100; z+=stride){
		float t[10];
		float t1[10];
		float t2[10];
		float t3[10];
		//fully connected forward
	  for(int i=0; i<10; i++){
	 	 t[i] = 0;
	 	 for(int j=0; j<784; j++) t[i] += weights[i*784+j]*inp[j+shuffled[z]*784];
	 	 t[i] += bias[i];
	  }
	  //softmax forward
	  float m = 0;
	  for(int i=0; i<10; i++){
	 	 if(t[i]>m) m = t[i];
	  }
	  float sum = 0;
	  for(int i=0; i<10; i++){
	 	 t1[i] = exp(t[i]-m);
	 	 sum += t1[i];
	  }
	  for(int i=0; i<10; i++) t1[i] = t1[i]/sum;
	  //cross entropy
	  for(int i=0; i<10; i++) t2[i] = 0;
	  t2[(int)labels[shuffled[z]]] = -1/t1[(int)labels[shuffled[z]]];
	  //softmax backprop
	  for(int i=0; i<10; i++){
	 	 t3[i] = 0;
	 	 for(int j=0; j<10; j++){
	 		 if(i == j) t3[i] += t2[j]*t1[i]*(1-t1[j]);
	 		 else t3[i] += t2[j]*t1[j]*(-t1[i]);
	 	 }
	  }
	  //fully connected backprop
	  for(int i=0; i<10; i++){
	 	 for(int j=0; j<784; j++) atomicAdd(&wd[i*784+j], t3[i]*inp[j+shuffled[z]*784]/100);
	 	 atomicAdd(&bd[i], t3[i]/100);
	  }
	}
}

int main(){
 int temp;
 unsigned char temp1;
 //import training images
 ifstream fin("train-images.idx3-ubyte", ios::binary);
 fin.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 float *training;
 hipMallocManaged(&training, 47040000*sizeof(float));
 for(size_t i = 0; i<47040000; i++){
	 fin.read(reinterpret_cast<char*>(&temp1), sizeof(unsigned char));
	 training[i] = float(temp1)/127.5-1;
 }
 fin.close();
 //import training labels
 ifstream fin2("train-labels.idx1-ubyte", ios::binary);
 fin2.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin2.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 unsigned char *trainingl;
 hipMallocManaged(&trainingl, 60000*sizeof(unsigned char));
 for(size_t i = 0; i<60000; i++){
	 fin2.read(reinterpret_cast<char*>(&trainingl[i]), sizeof(unsigned char));
 }
 fin2.close();
 //import testing images
 ifstream fin1("t10k-images.idx3-ubyte", ios::binary);
 fin1.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin1.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin1.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin1.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 float *testing;
 hipMallocManaged(&testing, 7840000*sizeof(float));
 for(size_t i = 0; i<7840000; i++){
	 fin1.read(reinterpret_cast<char*>(&temp1), sizeof(unsigned char));
	 testing[i] = float(temp1)/127.5-1;
 }
 fin1.close();
 //import testing labels
 ifstream fin3("t10k-labels.idx1-ubyte", ios::binary);
 fin3.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 fin3.read(reinterpret_cast<char*>(&temp), sizeof(temp));
 unsigned char * testingl;
 hipMallocManaged(&testingl, 10000*sizeof(unsigned char));
 for(size_t i = 0; i<10000; i++){
	 fin3.read(reinterpret_cast<char*>(&testingl[i]), sizeof(unsigned char));
 }
 fin3.close();
 float *weights;
 float *bias;
 float *wd;
 float *bd;
 hipMallocManaged(&weights, 7840*sizeof(float));
 hipMallocManaged(&bias, 10*sizeof(float));
 hipMallocManaged(&wd, 7840*sizeof(float));
 hipMallocManaged(&bd, 10*sizeof(float));
 std::normal_distribution<float> init;
 std::default_random_engine m_eng(100);
 //initialize weights to random numbers and bias to 0
 for(int i=0; i<7840; i++) weights[i] = init(m_eng)/28;
 for(int i=0; i<10; i++) bias[i] = 0;
 int *shuffled;
 hipMallocManaged(&shuffled, 60000*sizeof(int));
 for(int i=0; i<60000; i++) shuffled[i] = i;
 float *temper;
 hipMallocManaged(&temper, sizeof(float));
 forward<<<50,200>>>(testing, testingl, weights, bias, temper);
 hipDeviceSynchronize();
 cout<<"Initial: "<<*temper/10000<<endl;
 for(int e=0; e<30; e++){
	 //shuffle training data
	 random_shuffle(&shuffled[0], &shuffled[59999]);
	 for(int i=0;i<600; i++){
		 //set weight and bias derivatives to 0
		 for(int j=0; j<7840; j++) wd[j] = 0;
 		 for(int j=0; j<10; j++) bd[j] = 0;
		 //train the weights and bias
		 train<<<1,100>>>(training, trainingl, shuffled+100*i, wd, bd, weights, bias);
		 hipDeviceSynchronize();
		 //update weights and bias
		 for(int i=0; i<10; i++){
	 	 	 for(int j=0; j<784; j++) weights[i*784+j] -= .001*wd[i*784+j];
	 	 	 bias[i] -= .001*bd[i];
	 	  }
		}
		//run and print
	 forward<<<50,200>>>(testing, testingl, weights, bias, temper);
	 hipDeviceSynchronize();
	 cout<<"Epoch "<<e+1<<": "<<*temper/10000<<endl;
 }
 hipFree(shuffled);
 hipFree(trainingl);
 hipFree(testingl);
 hipFree(training);
 hipFree(testing);
 hipFree(weights);
 hipFree(bias);
 hipFree(wd);
 hipFree(bd);
 hipFree(temper);
 return 0;
}
